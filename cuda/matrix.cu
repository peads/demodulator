#include "hip/hip_runtime.h"
/*
 * This file is part of the demodulator distribution
 * (https://github.com/peads/demodulator).
 * with code originally part of the misc_snippets distribution
 * (https://github.com/peads/misc_snippets).
 * Copyright (c) 2023 Patrick Eads.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */
#include "nvidia.cuh"

__global__
void fmDemod(const uint8_t *buf, const uint32_t len, const float gain, float *result) {

    uint32_t i;
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t step = blockDim.x * gridDim.x;
    float a, b, c, d, ac, bd, zr, zj;

    for (i = index; i < len; i += step) {

        a = __int2float_rn(buf[i] + buf[i + 2] - 254);
        b = __int2float_rn(254 - buf[i + 1] - buf[i + 3]);

        c = __int2float_rn(buf[i + 4] + buf[i + 6] - 254);
        d = __int2float_rn(buf[i + 5] + buf[i + 7] - 254);

        ac = a * c;
        bd = b * d;
        zr = ac - bd;
        zj = (a + b) * (c + d) - (ac + bd);

        zj = 64.f * zj;
        zr = zj * __frcp_rn(23.f * zr + 41.f);

        result[i >> 2] = isnan(zr) ? 0.f : gain ? gain * zr : zr;
    }
}

extern "C" void *processMatrix(void *ctx) {

    auto *args = static_cast<consumerArgs *>(ctx);
    float *dResult;
    uint8_t *dBuf;
    float *hResult;

    hipMalloc(&dBuf, DEFAULT_BUF_SIZE);
    hipMalloc(&dResult, (DEFAULT_BUF_SIZE >> 2) * sizeof(float));
    hipHostMalloc(&hResult, (DEFAULT_BUF_SIZE >> 2) * sizeof(float));

    while (!args->exitFlag) {

        sem_wait(&args->full);
        pthread_mutex_lock(&args->mutex);
        hipMemcpy(dBuf, args->buf, DEFAULT_BUF_SIZE, hipMemcpyHostToDevice);
        pthread_mutex_unlock(&args->mutex);
        sem_post(&args->empty);

        hipDeviceSynchronize();
        fmDemod<<<GRIDDIM, BLOCKDIM>>>(dBuf, DEFAULT_BUF_SIZE, args->gain, dResult);

        hipDeviceSynchronize();
        hipMemcpy(hResult,
                dResult,
                (DEFAULT_BUF_SIZE >> 2) * sizeof(float),
                hipMemcpyDeviceToHost);

        hipDeviceSynchronize();
        fwrite(hResult, sizeof(float), DEFAULT_BUF_SIZE >> 2, args->outFile);
    }

    hipHostFree(args->buf);
    hipHostFree(hResult);
    hipFree(dBuf);
    hipFree(dResult);

    return nullptr;
}

extern "C" void allocateBuffer(void **buf, const size_t len) {

    hipHostMalloc(buf, len);
}

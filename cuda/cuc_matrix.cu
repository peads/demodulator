#include "hip/hip_runtime.h"
/*
 * This file is part of the demodulator distribution
 * (https://github.com/peads/demodulator).
 * with code originally part of the misc_snippets distribution
 * (https://github.com/peads/misc_snippets).
 * Copyright (c) 2023 Patrick Eads.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */
#include <hip/hip_complex.h>
#include "nvidia.cuh"

__global__
void fmDemod(const uint8_t *buf, const uint32_t len, float *result) {

    uint32_t i;
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t step = blockDim.x * gridDim.x;
    hipComplex a, b, z;

    for (i = index; i < len; i += step) {

        a = {
            __int2float_rn(buf[i] + buf[i + 2] - 254),
            __int2float_rn(254 - buf[i + 1] - buf[i + 3])};
        b = {
            __int2float_rn(buf[i + 4] + buf[i + 6] - 254),
            __int2float_rn(buf[i + 5] + buf[i + 7] - 254)};
        z = hipCmulf(a, b);

        z.y = __fmul_rn(64.f, z.y);
        z.x = __fmul_rn(z.y, __frcp_rn(__fmaf_rn(23.f, z.x, 41.f)));
        result[i >> 2] = isnan(z.x) ? 0.f : z.x; // delay line
//        result[i >> 2] = atan2f(z.y, z.x);
    }
}

extern "C" int processMatrix(FILE *__restrict__ inFile,
                             const uint8_t mode,
                             float gain,
                             void *__restrict__ outFile) {

    int exitFlag = mode != 1;
    uint8_t *dBuf;
    uint8_t *hBuf;
    float *dResult;
    float *hResult;
    size_t readBytes;
    const uint8_t isGain = fabsf(1.f - gain) > GAIN_THRESHOLD;


    hipHostMalloc(&hBuf, sizeof(hipComplex) * (DEFAULT_BUF_SIZE >> 1));
    hipMalloc(&dBuf, sizeof(hipComplex) * (DEFAULT_BUF_SIZE >> 1));
    hipHostMalloc(&hResult, (DEFAULT_BUF_SIZE >> 2) * OUTPUT_ELEMENT_BYTES);
    hipMalloc(&dResult, (DEFAULT_BUF_SIZE >> 2) * OUTPUT_ELEMENT_BYTES);

    hBuf[0] = 0;
    hBuf[1] = 0;

    while (!exitFlag) {

        readBytes = fread(hBuf + 2, 1, DEFAULT_BUF_SIZE - 2, inFile);

        if ((exitFlag = ferror(inFile))) {
            perror(nullptr);
            break;
        } else if (feof(inFile)) {
            exitFlag = EOF;
        }

        hipMemcpy(dBuf, hBuf, DEFAULT_BUF_SIZE, hipMemcpyHostToDevice);

        fmDemod<<<GRIDDIM, BLOCKDIM>>>(dBuf, readBytes + 2, dResult);

        hipMemcpy(hResult,
            dResult,
            ((readBytes + 2) >> 2) * OUTPUT_ELEMENT_BYTES,
            hipMemcpyDeviceToHost);

        fwrite(hResult, OUTPUT_ELEMENT_BYTES, (readBytes + 2) >> 2, (FILE *) outFile);
    }

    hipHostFree(hBuf);
    hipHostFree(hResult);
    hipFree(dBuf);
    hipFree(dResult);
    return exitFlag;
}
